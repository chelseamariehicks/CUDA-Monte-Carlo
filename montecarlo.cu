#include "hip/hip_runtime.h"
/**********************************************************************************
 * Name: Chelsea Marie Hicks
 * 
 * Description: 
 *
 * Resources include: CS475 documentation 
***********************************************************************************/

#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <malloc.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "helper_functions.h"
#include "hip/hip_runtime_api.h"

//Print debugging messages
#ifndef DEBUG
#define DEBUG       false
#endif


//Set number of traisl in monte carlo simulation, multiples of 1024
#ifndef NUMTRIALS
#define NUMTRIALS           (1024 * 2)
#endif

#ifndef BLOCKSIZE
#define BLOCKSIZE           16
#endif

#define NUMBLOCKS           (NUMTRIALS / BLOCKSIZE)

//Ranges for the random numbers:
const float GMIN =	20.0;	// ground distance in meters
const float GMAX =	30.0;	// ground distance in meters
const float HMIN =	10.0;	// cliff height in meters
const float HMAX =	40.0;	// cliff height in meters
const float DMIN  =	10.0;	// distance to castle in meters
const float DMAX  =	20.0;	// distance to castle in meters
const float VMIN  =	30.0;	// intial cnnonball velocity in meters / sec
const float VMAX  =	50.0;	// intial cnnonball velocity in meters / sec
const float THMIN = 70.0;	// cannonball launch angle in degrees
const float THMAX =	80.0;	// cannonball launch angle in degrees

const float GRAVITY =	-9.8;	// acceleraion due to gravity in meters / sec^2
const float TOL = 5.0;		    // tolerance in cannonball hitting the castle in meters
				                // castle is destroyed if cannonball lands between d-TOL and d+TOL

//Helper Functions
float Ranf(float low, float high) {
    float r = (float) rand();
    float t = r / (float) RAND_MAX;

    return low + t * (high - low);
}

int Ranf(int ilow, int ihigh) {
    float low = (float) ilow;
    float high = ceil((float) ihigh);

    return (int) Ranf(low, high);
}

void TimeOfDaySeed() {
    struct tm y2k = { 0 };
    y2k.tm_hour = 0;
    y2k.tm_min = 0;
    y2k.tm_sec = 0;
    y2k.tm_year = 100;
    y2k.tm_mon = 0;
    y2k.tm_mday = 1;

    time_t timer;
    time(&timer);
    double seconds = difftime(timer, mktime(&y2k));
    unsigned int seed = (unsigned int) (1000.*seconds); //milli
    srand(seed);
}

void CudaCheckError() {
    hipError_t e = hipGetLastError();
    if (e != hipSuccess) {
        fprintf(stderr, "CUDA failure %s:%d: '%s'\n", __FILE__, __LINE__, hipGetErrorString(e));
    }
}

//Function prototypes:
float       Ranf(float, float);
int         Ranf(int, int);
void        TimeOfDaySeed();

// degrees-to-radians -- callable from the device:
__device__ float Radians(float d) {
    return (M_PI/180.f) * d;
}

// the kernel:
__global__ void MonteCarlo( float *dvs, float *dths, float *dgs, float *dhs, float *dds, int *dhits ) {
    unsigned int gid = blockIdx.x * blockDim.x + threadIdx.x;
        
    // randomize everything:
    float v   = dvs[gid];
    float thr = Radians(dths[gid]);
    float vx  = v * cos(thr);
    float vy  = v * sin(thr);
    float  g  =  dgs[gid];
    float  h  =  dhs[gid];
    float  d  =  dds[gid];

    int numHits = 0;

    //See if the ball doesn't even reach the cliff
    float t = (-2. * vy) / GRAVITY;
    float x = vx * t;

    if(x <= g) {
        if(DEBUG) {
            //fprintf(stderr, "Ball doesn't even reach the cliff\n");
        }
    }
    else {
        //See if the ball hits the cliff face
        float t = g / vx;
        float y = vy * t + (0.5 * GRAVITY * (t*t));
        if(y <= h) {
            if(DEBUG) {
                //fprintf(stderr, "Ball hits the cliff face\n");
            }
        }
        else {
            //Ball hits the upper deck
            //the time solution for this is quadratic equation of the form:
            //at^2 + bt + c = 0
            //where 'a' multiplies time^2
            //      'b' multiples time
            //      'c' is a constant
            float a = 0.5 * GRAVITY;
            float b = vy;
            float c = -h;
            float disc = b * b - 4.f * a * c; //quadratic formula discriminant

            //Ball doesn't go as high as the upper deck:
            //this should "never happen"...
            if(disc < 0.) {
                if(DEBUG) {
                    //fprintf(stderr, "Ball doesn't reach upper deck.\n");
                    //exit(1); //something is wrong...
                }
            }
            
            //Ball successfully hits the ground above the cliff:
            //get the intersection:
            disc = sqrtf(disc);
            float t1 = (-b + disc) / (2.f * a);   //time to intersect high ground
            float t2 = (-b - disc) / (2.f * a);   //time to intersect high ground

            //only care about the second intersection
            float tmax = t1;
            if (t2 > t1) {
                tmax = t2;
            }

            //How far does the ball land horizontally from the edge of the cliff?
            float upperDist = vx * tmax - g;

            //See if the ball hits the castle
            if(fabs(upperDist - d) > TOL) {
                if(DEBUG) {
                    //fprintf(stderr, "Misses the castle at upperDist = %8.3f\n", upperDist);
                }
            }
            else {
                if(DEBUG) {
                    //fprintf(stderr, "Hits the castle at upperDist = %8.3f\n", upperDist);
                }
                numHits = 1;
            } 
        }
    }
    dhits[gid] = numHits;
}

//These two #defines are just to label things
//Other than that, they do nothing:
#define IN
#define OUT

int main(int argc, char* argv[]) {
    TimeOfDaySeed( );

    int dev = findCudaDevice(argc, (const char **)argv);

    //Better to define these here so that the rand() calls don't get into the thread timing:
    float *hvs   = new float [NUMTRIALS];
    float *hths  = new float [NUMTRIALS];
    float *hgs   = new float [NUMTRIALS];
    float *hhs   = new float [NUMTRIALS];
    float *hds   = new float [NUMTRIALS];
    int   *hhits = new int   [NUMTRIALS];

    //Fill in the random value arrays
    for(int n = 0; n < NUMTRIALS; n++) {
        hvs[n] = Ranf(VMIN, VMAX);
        hths[n] = Ranf(THMIN, THMAX);
        hgs[n] = Ranf(GMIN, GMAX);
        hhs[n] = Ranf(HMIN, HMAX);
        hds[n] = Ranf(DMIN, DMAX);
    }

    //Allocate device memory:
    float *dvs, *dths, *dgs, *dhs, *dds;
    int   *dhits;

    hipMalloc( &dvs,   NUMTRIALS*sizeof(float) );
    hipMalloc( &dths,  NUMTRIALS*sizeof(float) );
    hipMalloc( &dgs,   NUMTRIALS*sizeof(float) );
    hipMalloc( &dhs,   NUMTRIALS*sizeof(float) );
    hipMalloc( &dds,   NUMTRIALS*sizeof(float) );
    hipMalloc( &dhits, NUMTRIALS*sizeof(int) );
    CudaCheckError();

    //Copy host memory to the device:
    hipMemcpy( dvs,  hvs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dths, hths, NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dgs,  hgs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dhs,  hhs,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dds,  hds,  NUMTRIALS*sizeof(float), hipMemcpyHostToDevice );
    CudaCheckError();

    //Setup the execution parameters:
    dim3 grid(NUMBLOCKS, 1, 1);
    dim3 threads(BLOCKSIZE, 1, 1);

    //Allocate cuda events that we'll use for timing:
    hipEvent_t start, stop;
    hipEventCreate( &start );
    hipEventCreate( &stop  );
    CudaCheckError();

    //Let the gpu go quiet:
    hipDeviceSynchronize( );

    //Record the start event:
    hipEventRecord( start, NULL );
    CudaCheckError();

    //Execute the kernel:
    MonteCarlo<<< grid, threads >>>(IN dvs, IN dths, IN dgs, IN dhs, IN dds, OUT dhits);

    //Record the stop event:
    hipEventRecord( stop, NULL );
    CudaCheckError();

    //Wait for the stop event to complete:
    hipDeviceSynchronize( );
    hipEventSynchronize( stop );
    CudaCheckError();

    float msecTotal = 0.0f;
    hipEventElapsedTime( &msecTotal, start, stop );
    CudaCheckError();

    //Compute and print the performance
    double totalSecs = 0.001 * (double) msecTotal;
    double trialsPerSec = (float) NUMTRIALS / totalSecs;
    double megaTrialsPerSec = trialsPerSec / 1000000.;
    fprintf(stderr, "Number of trials = %10d, MegaTrials/second = %10.4lf\n", NUMTRIALS, megaTrialsPerSec);

    //Copy result from the device to the host:
    hipMemcpy(hhits, dhits, NUMTRIALS * sizeof(int), hipMemcpyDeviceToHost);
    CudaCheckError();

    // add up the hhits[ ] array: :
	int numHits = 0;
    for(int i = 0; i < NUMTRIALS; i++) {
        numHits += hhits[i];
    }

    // compute and print the probability:
    float probability = 100.f * (float) numHits / (float) NUMTRIALS;
    fprintf(stderr, "Probability = %6.3f %%\n", probability);

    //Clean up host memory:
    delete [ ] hvs;
    delete [ ] hths;
    delete [ ] hgs;
    delete [ ] hhs;
    delete [ ] hds;
    delete [ ] hhits;

    //Clean up device memory:
    hipFree(dvs);
    hipFree(dths);
    hipFree(dgs);
    hipFree(dhs);
    hipFree(dds);
    hipFree(dhits);
    CudaCheckError();

    return 0;
}
